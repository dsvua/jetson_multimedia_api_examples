#include "hip/hip_runtime.h"
#include "histogram.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include "rawImageSaver.h"
#include <iostream>


__global__ void cudaDumpImage(hipSurfaceObject_t surface_left, hipSurfaceObject_t surface_right, uint8_t* d_image, unsigned int width, unsigned int height){
    // global position and size
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x<width && y < height) {
        d_image[y*width+x]              = surf2Dread<uint8_t>(surface_left, x, y);
        d_image[y*width+x+height*width] = surf2Dread<uint8_t>(surface_right, x, y);

        // printing for debugging purpose, it should pring any number but not 0
        if (x == 1000 && y == 300){
            printf("surf2Dread at 1000x300 left: %3u, right: %3u \n", surf2Dread<uint8_t>(surface_left, x, y), surf2Dread<uint8_t>(surface_right, x, y));
        }
    }
}

void checkCudaError(hipError_t err, const char * message){
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to %s (error code %03u %s)!\n",
                         message, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }    
}

void imageDumper(hipSurfaceObject_t surface_left, hipSurfaceObject_t surface_right, unsigned int width, unsigned int height, unsigned int frameNumber)
{
    // Set flag to enable zero copy access
    hipError_t err = hipSuccess;

    dim3 block(8, 8);
    dim3 grid(width/block.x+1,height/block.y+1);

    printf("Picture size is %u x %u \n", width, height);
    printf("hipMallocManaged image memory\n");
    uint8_t *image, *d_image;
    size_t mallocSize = width*height*sizeof(uint8_t)*2;
    image = (uint8_t*)malloc(mallocSize); // multiply by 2 as we have 2 images
    // err = hipMallocManaged(&image,width*height*sizeof(uint8_t));
    // checkCudaError(err, "hipMallocManaged");
 
    err = hipMalloc(&d_image, mallocSize);
    checkCudaError(err, "hipMalloc");

    printf("Calling cudaDumpImage kernel...\n");
    cudaDumpImage<<<grid, block>>>(surface_left, surface_right, d_image, width, height);
    err = hipGetLastError();
    checkCudaError(err, "cudaDumpImage kernel");

    err = hipMemcpy(image, d_image, mallocSize, hipMemcpyDeviceToHost);
    checkCudaError(err, "hipMemcpy");
    // for(int i = 13120; i < 13320; i++) printf("results[%1u] %3u \n", i, image[i]);
    printf("results[%1u] %3u \n", 13132, image[13132]);

    err = hipDeviceSynchronize();
    checkCudaError(err, "hipDeviceSynchronize");

    printf("Dumping frame to image %d \n", frameNumber);

    char filename_bin[256];
    sprintf(filename_bin, "output%03u.bin", frameNumber);
    rawImageSaver(width, height*2, image, filename_bin); // height*2 as we have two images

    printf("Freeing d_image memory\n");
    hipFree(d_image);
    printf("Freeing image memory\n");
    delete image;
}
